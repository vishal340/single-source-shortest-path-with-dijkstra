
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<string>
#include<cstdlib>
#include<cstring>
#include<vector>
#include<iterator>
#include<ctime>
#include<limits>
#include<list>
#include<algorithm>

using namespace std;

struct info_edge     
{
int vertex1,vertex2;
int edge;
};

//This function to extract data from file
void extract_data(info_edge &info,char* str)
{
int i,m=0;
int vertex[2];
int weight;
vertex[0]=0;
vertex[1]=0;
for(i=2;i<strlen(&str[0])-1;i++)
{
if(str[i]!=' ')
{
vertex[m]*=10;
vertex[m]+=(int)str[i]-48;
}
else if(m<1)
{
m++;
}
else
break;
}
weight=0;
i++;
while(i<strlen(&str[0]))
{
weight*=10;
weight+=(int)str[i]-48;
i++;
}
info.vertex1=--vertex[0];
info.vertex2=--vertex[1];
info.edge=weight;
}



//This is the kernel
__global__ void sssp(pair<int,int>* adjacency,int* dist,int* change,int* count)
{
int n=blockDim.x*blockIdx.x + threadIdx.x;
int n1=n*10;
//if(n%10000==0)
//printf("%d\t",count[n]);
if(dist[n]!=2147483647)
for(int i=0;i<count[n];i++)
{
if(dist[adjacency[n1+i].first] > adjacency[n1+i].second + dist[n])
{
change[0]=1;
dist[adjacency[n1+i].first] = adjacency[n1+i].second + dist[n];
}
}
}

int main(int argc,char** argv)
{
struct timespec start,finish;
ifstream in(argv[2]);
string str;
for(int i=0;i<4;i++)
getline(in,str);

getline(in,str);
int vertex=0;
int edge=0;
int i=5;

//these 2 while loop for finding number of veritices and edges
while(str[i]!=' ')
{
vertex*=10;
vertex+=(int)str[i]-48;
i++;
}
i++;
while(i<strlen(&str[0])-1)
{
edge*=10;
edge+=(int)str[i]-48;
i++;
}
vertex++;



pair<int,int>* adjacency1=new pair<int,int>[vertex*10];//datasructure of graph
int* count1=new int[vertex];

for(i=0;i<2;i++)
getline(in,str);
int* dist1=new int[vertex];

int source=atoi(argv[1]);
dist1[source]=0;

for(int i=0;i<vertex;i++)
{
count1[i]=0;
if(i!=source)
dist1[i]=std::numeric_limits<int>::max();
}
info_edge info;
pair<int,int> sub_info;


while(!in.eof())    		//loop on each line of file
{
getline(in,str);
extract_data(info,&str[0]);
sub_info.first=info.vertex2;
sub_info.second=info.edge;
adjacency1[info.vertex1*10+count1[info.vertex1]]=sub_info;
count1[info.vertex1]++;
}


pair<int,int>* adjacency;
int *dist,*count;
int* change1=new int[1];
int* change;

hipMalloc((void**)&adjacency,sizeof(int)*2*vertex*10);
hipMalloc((void**)&dist,sizeof(int)*vertex);
hipMalloc((void**)&count,sizeof(int)*vertex);
hipMemcpy(adjacency,adjacency1,sizeof(int)*2*vertex*10,hipMemcpyHostToDevice);
hipMemcpy(dist,dist1,sizeof(int)*vertex,hipMemcpyHostToDevice);
hipMemcpy(count,count1,sizeof(int)*vertex,hipMemcpyHostToDevice);
dim3 dimBlock(512);
dim3 dimGrid(3693);

int iteration=0;
clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start);


do
{
change1[0]=0;
hipMalloc((void**)&change,sizeof(int));
hipMemcpy(change,change1,sizeof(int),hipMemcpyHostToDevice);

sssp<<<dimGrid,dimBlock>>>(adjacency,dist,change,count);

hipMemcpy(change1,change,sizeof(int),hipMemcpyDeviceToHost);
iteration++;
}while(change1[0]==1);

clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&finish);
hipMemcpy(dist1,dist,sizeof(int)*vertex,hipMemcpyDeviceToHost);



cout<<"no.of iterations: "<<iteration<<"\n";
cout<<"Time taken\t"<<(finish.tv_sec-start.tv_sec)+(finish.tv_nsec-start.tv_nsec)/1e09<<"\n";
ofstream out("output.txt");
for(int i=0;i<vertex-1;i++)
out<<i+1<<"\t"<<dist1[i]<<"\n";
return 0;
}
