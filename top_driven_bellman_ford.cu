
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<string>
#include<cstdlib>
#include<cstring>
#include<vector>
#include<iterator>
#include<ctime>
#include<limits>

using namespace std;

struct info_edge
{
int vertex1,vertex2;
int weight;
};

void extract_data(vector<info_edge> &adjacency,char* str)
{
int i,n=1,m=0;
int vertex[2];
int weight;
vertex[0]=0;
vertex[1]=0;
for(i=2;i<strlen(&str[0])-1;i++)
{
if(str[i]!=' ')
{
vertex[m]*=10;
vertex[m]+=(int)str[i]-48;
}
else if(m<1)
{
m++;
}
else
break;
}
info_edge edge;
if(vertex[0]<vertex[1])
{
edge.vertex1=--vertex[0];
edge.vertex2=--vertex[1];
}
else
return;
weight=0;
i++;
while(i<strlen(&str[0]))
{
weight*=10;
weight+=(int)str[i]-48;
i++;
}
edge.weight=weight;
adjacency.push_back(edge);
}

void sssp(vector<info_edge>adjacency,int* dist)
{
vector<info_edge>::iterator iter;
bool change=true;
int i=1;
while(change)
{
change=false;
for(iter=adjacency.begin();iter<adjacency.end();iter++)
{
if(dist[iter->vertex1] != dist[iter->vertex2])
{
int t=dist[iter->vertex1]-dist[iter->vertex2];
if(t>iter->weight)
{
change=true;
dist[iter->vertex1]=dist[iter->vertex2]+iter->weight;
}
else if(-t>iter->weight)
{
change=true;
dist[iter->vertex2]=dist[iter->vertex1]+iter->weight;
}
}
}
i++;
}
cout<<"Number of iterations :"<<--i<<"\n";
}

int main(int argc,char** argv)
{
struct timespec start,finish;
ifstream in(argv[2]);
string str;
int vertex;
int edge;
vector<info_edge> adjacency;
int* dist;
for(int i=0;i<4;i++)
{
getline(in,str);
}

getline(in,str);
vertex=0;
edge=0;
int i=5;
while(str[i]!=' ')
{
vertex*=10;
vertex+=(int)str[i]-48;
i++;
}
i++;
while(i<strlen(&str[0])-1)
{
edge*=10;
edge+=(int)str[i]-48;
i++;
}
for(i=0;i<2;i++)
getline(in,str);

dist=new int[vertex];
for(int i=0;i<vertex;i++)
dist[i]=std::numeric_limits<int>::max();
while(!in.eof())
{
getline(in,str);
extract_data(adjacency,&str[0]);
}
int source=atoi(argv[1]);
dist[source]=0;

clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start);
sssp(adjacency,dist);
clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&finish);
cout<<"Time taken\t"<<(finish.tv_sec-start.tv_sec)+(finish.tv_nsec-start.tv_nsec)/1e09<<"\n";
ofstream out("output1.txt");
for(int i=0;i<vertex;i++)
out<<i+1<<"\t"<<dist[i]<<"\n";
return 0;
}
